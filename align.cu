#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>

#define BLOCK_SIZE 128

#ifdef SEQ_STYLE
__constant__ char seqB[64000];
#else
//__constant__ float seqB[128][4];
__constant__ float seqB[1600][4];
#endif

texture<float, 2, hipReadModeElementType> tex;

//__device__ float sdata[256];

char* d_seqA;
float* d_scores;
float* d_row;
int currentDeviceNum = 0;

__device__ void calc(char& a, float* b, float& leftVal, float& diag, float& above, float& max, float& GAP)
{
#ifdef DEBUG
	printf("%.2f %.2f %.2f %.2f against %c\n",b[0],b[1],b[2],b[3],a);
#endif
	// Gets the alignment score from the "texture" (score matrix) with a and b
	max = tex2D(tex, 'a', a) * b[0];
	max += tex2D(tex, 'c', a) * b[1];
	max += tex2D(tex, 'g', a) * b[2];
	max += tex2D(tex, 't', a) * b[3];


    float score2 = leftVal + GAP;

    max += diag;
    float score3 = above + GAP;

    if ( score2 > max )
    {
        max = score2;
    }
    if ( score3 > max )
    {
        max = score3;
    }
}

__device__ void calc(char& a, char& b, float& leftVal, float& diag, float& above, float& max, float& GAP)
{

	// Gets the alignment score from the "texture" (score matrix) with a and b
	max = tex2D(tex, a, b);

    float score2 = leftVal + GAP;

    max += diag;
    float score3 = above + GAP;

    if ( score2 > max )
    {
        max = score2;
    }
    if ( score3 > max )
    {
        max = score3;
    }

//    printf("Comparing %c with %c, left=%f diag=%f above=%f max=%f gap=%f\n",a,b,leftVal,diag,above,max,GAP);

}

__global__ void align(char* dSeqA, float* dScores, float* dRow, int seqALen, int seqBLen, float GAP)
{
    int tid = threadIdx.x;
    int blockRowStart = (blockDim.x * seqBLen) * blockIdx.x;
    int blockSeqAStart = (blockDim.x * seqALen) * blockIdx.x;
    float max;
#ifdef DEBUG
    if(blockIdx.x != 0)
    	printf("[%d] blockDim.x=%d, blockIdx.x=%d, Aligning seqA=%s\n",tid,blockDim.x,blockIdx.x, dSeqA);
#endif

    float v0, v1, v2, v3, v4, v5, v6, v7, v8, v9, v10, v11, v12, v13, v14, v15;

    max = GAP;
    int i;
    for ( i = 0; i < seqBLen; ++i ) {
        dRow[blockRowStart + i * blockDim.x + tid] = max;
        max += GAP;
    }

    float leftVal, above, diag;

    char a;

    int remaining = seqBLen;
    float loop = 0.0f;
    float oldLeft = 0;
    int idx;
    while ( remaining > 15 )
    {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;
        v9 = GAP * 10.0f + addGap;
        v10 = GAP * 11.0f + addGap;
        v11 = GAP * 12.0f + addGap;
        v12 = GAP * 13.0f + addGap;
        v13 = GAP * 14.0f + addGap;
        v14 = GAP * 15.0f + addGap;
        v15 = GAP * 16.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            if ( loop > 0 )
            //if ( row > 0 )
                leftVal = dRow[blockRowStart + row * blockDim.x + tid];
            else
                leftVal = (row+1) * GAP;

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];
#ifdef DEBUG
            if(blockIdx.x != 0)
            	printf("at:%d(%c=%d) ",tid + row * blockDim.x + blockSeqAStart,a,a);
#endif

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            leftVal = max;

            diag = above;
            v0 = max;

            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v1 = max;

            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v2 = max;


            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v3 = max;

            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v4 = max;

            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v5 = max;

            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v6 = max;

            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v7 = max;

            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v8 = max;

            above = v9;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v9 = max;

            above = v10;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v10 = max;

            above = v11;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v11 = max;

            above = v12;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v12 = max;

            above = v13;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v13 = max;

            above = v14;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v14 = max;

            above = v15;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            leftVal = max;
            diag = above;
            v15 = max;

            dRow[blockRowStart + row * blockDim.x + tid] = max;

        }
        remaining -= 16;
        loop += 1.0f;
    }
    if ( remaining == 0 ) {
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;
    }


    if ( remaining == 1 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;

        idx = (int)(loop * 16.0f);
        for ( int row = 0; row < seqALen; ++row ) {

            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;
        }

        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;
    }


    else if ( remaining == 2 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;
    }


    else if ( remaining == 3 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }

    else if ( remaining == 4 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }

    else if ( remaining == 5 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 6 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 7 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 8 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }

    else if ( remaining == 9 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;

            leftVal = max;
            diag = above;
            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v8 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }

    else if ( remaining == 10 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;
        v9 = GAP * 10.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;

            leftVal = max;
            diag = above;
            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v8 = max;

            leftVal = max;
            diag = above;
            above = v9;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v9 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 11 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;
        v9 = GAP * 10.0f + addGap;
        v10 = GAP * 11.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;

            leftVal = max;
            diag = above;
            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v8 = max;

            leftVal = max;
            diag = above;
            above = v9;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v9 = max;

            leftVal = max;
            diag = above;
            above = v10;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v10 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 12 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;
        v9 = GAP * 10.0f + addGap;
        v10 = GAP * 11.0f + addGap;
        v11 = GAP * 12.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;

            leftVal = max;
            diag = above;
            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v8 = max;

            leftVal = max;
            diag = above;
            above = v9;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v9 = max;

            leftVal = max;
            diag = above;
            above = v10;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v10 = max;

            leftVal = max;
            diag = above;
            above = v11;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v11 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 13 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;
        v9 = GAP * 10.0f + addGap;
        v10 = GAP * 11.0f + addGap;
        v11 = GAP * 12.0f + addGap;
        v12 = GAP * 13.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;

            leftVal = max;
            diag = above;
            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v8 = max;

            leftVal = max;
            diag = above;
            above = v9;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v9 = max;

            leftVal = max;
            diag = above;
            above = v10;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v10 = max;

            leftVal = max;
            diag = above;
            above = v11;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v11 = max;

            leftVal = max;
            diag = above;
            above = v12;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v12 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 14 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;
        v9 = GAP * 10.0f + addGap;
        v10 = GAP * 11.0f + addGap;
        v11 = GAP * 12.0f + addGap;
        v12 = GAP * 13.0f + addGap;
        v13 = GAP * 14.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;

            leftVal = max;
            diag = above;
            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v8 = max;

            leftVal = max;
            diag = above;
            above = v9;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v9 = max;

            leftVal = max;
            diag = above;
            above = v10;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v10 = max;

            leftVal = max;
            diag = above;
            above = v11;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v11 = max;

            leftVal = max;
            diag = above;
            above = v12;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v12 = max;

            leftVal = max;
            diag = above;
            above = v13;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v13 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


    else if ( remaining == 15 ) {
        float addGap = (GAP * loop * 16.0f);
        v0 = GAP + addGap;
        v1 = GAP * 2.0f + addGap;
        v2 = GAP * 3.0f + addGap;
        v3 = GAP * 4.0f + addGap;
        v4 = GAP * 5.0f + addGap;
        v5 = GAP * 6.0f + addGap;
        v6 = GAP * 7.0f + addGap;
        v7 = GAP * 8.0f + addGap;
        v8 = GAP * 9.0f + addGap;
        v9 = GAP * 10.0f + addGap;
        v10 = GAP * 11.0f + addGap;
        v11 = GAP * 12.0f + addGap;
        v12 = GAP * 13.0f + addGap;
        v13 = GAP * 14.0f + addGap;
        v14 = GAP * 15.0f + addGap;

        for ( int row = 0; row < seqALen; ++row ) {
            idx = (int)(loop * 16.0f);
            above = v0;
            leftVal = dRow[blockRowStart + row * blockDim.x + tid];

            a = dSeqA[tid + row * blockDim.x + blockSeqAStart];

            calc(a, seqB[idx++], leftVal, oldLeft, above, max, GAP);
            oldLeft = leftVal;
            v0 = max;

            leftVal = max;
            diag = above;
            above = v1;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v1 = max;

            leftVal = max;
            diag = above;
            above = v2;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v2 = max;

            leftVal = max;
            diag = above;
            above = v3;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v3 = max;

            leftVal = max;
            diag = above;
            above = v4;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v4 = max;

            leftVal = max;
            diag = above;
            above = v5;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v5 = max;

            leftVal = max;
            diag = above;
            above = v6;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v6 = max;

            leftVal = max;
            diag = above;
            above = v7;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v7 = max;

            leftVal = max;
            diag = above;
            above = v8;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v8 = max;

            leftVal = max;
            diag = above;
            above = v9;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v9 = max;

            leftVal = max;
            diag = above;
            above = v10;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v10 = max;

            leftVal = max;
            diag = above;
            above = v11;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v11 = max;

            leftVal = max;
            diag = above;
            above = v12;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v12 = max;

            leftVal = max;
            diag = above;
            above = v13;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v13 = max;

            leftVal = max;
            diag = above;
            above = v14;
            calc(a, seqB[idx++], leftVal, diag, above, max, GAP);
            v14 = max;
        }
        dScores[blockDim.x * blockIdx.x + tid] = max;
        return;

    }


}

/**
 * This is the reduceSum function as described in the Project Requirements.
 * Will be needed to sum scores over each alignment
 */
#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif
__global__ void reduceSum1(float *g_idata, float *g_odata)
{
	extern __shared__ float sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;

    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
#ifdef __DEVICE_EMULATION__
    fprintf(stderr,"[%d] trying to sync...%f\n",tid,sdata[tid]);
#endif
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>32; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        sdata[tid] += sdata[tid + 32]; EMUSYNC;
        sdata[tid] += sdata[tid + 16]; EMUSYNC;
        sdata[tid] += sdata[tid +  8]; EMUSYNC;
        sdata[tid] += sdata[tid +  4]; EMUSYNC;
        sdata[tid] += sdata[tid +  2]; EMUSYNC;
        sdata[tid] += sdata[tid +  1]; EMUSYNC;
#ifdef __DEVICE_EMULATION__
        fprintf(stderr,"[%d] data is %f\n",tid,sdata[tid]);
#endif
    }

    // write result for this block to global mem
    if (tid == 0) *g_odata += sdata[0];
}

extern "C" bool setCudaDevice(int device)
{
    if ( hipSetDevice(device) != hipSuccess )
        return false;
    currentDeviceNum = device;
    return true;
}

extern "C" const char* initScoreMatrixTexture(float score_matrix[128][128])
{

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, 128, 128);
    hipMemcpyToArray(cuArray, 0, 0, score_matrix, 128*128*sizeof(float), hipMemcpyHostToDevice);

    // Set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode	= hipFilterModePoint;
    tex.normalized	= false;

    // Bind the array to the texture
    hipBindTextureToArray(tex, cuArray, channelDesc);
    if ( hipGetLastError() != hipSuccess )
  	return hipGetErrorString(hipGetLastError());
    return NULL;
}

extern "C" const char* setMatchSequence(char* matchSeq, int matchLen)
{
    //copy match sequence to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(seqB), matchSeq, matchLen, 0, hipMemcpyHostToDevice);
    if ( hipGetLastError() != hipSuccess )
  	return hipGetErrorString(hipGetLastError());
    return NULL;
}

extern "C" const char* setRead(float* matchSeq, int matchLen)
{
    //copy match sequence to constant memory
										// Need to copy one for each character (4)
    hipMemcpyToSymbol((char*)seqB, (char*)matchSeq, matchLen*4*sizeof(float), 0, hipMemcpyHostToDevice);

    if ( hipGetLastError() != hipSuccess )
    	return hipGetErrorString(hipGetLastError());

    return NULL;
}

extern "C" const char* initSequences(int seqALen, int blockSize, int numBlocks)
{
    hipMalloc( (void**)&d_seqA, numBlocks * blockSize * seqALen );
    if ( hipGetLastError() != hipSuccess )
  	return hipGetErrorString(hipGetLastError());
    return NULL;
}

extern "C" const char* setSequences(char* seqA, int start, int bytes)
{
    hipMemcpy( (void*)d_seqA, (void*)&(seqA[start]), bytes, hipMemcpyHostToDevice );
    if ( hipGetLastError() != hipSuccess )
  	return hipGetErrorString(hipGetLastError());
    return NULL;
}

extern "C" const char* copyScores(float* scores, int start, int bytes)
{
    hipMemcpy( (void*)&(scores[start]), (void*)d_scores, bytes, hipMemcpyDeviceToHost );
    if ( hipGetLastError() != hipSuccess )
    	return hipGetErrorString(hipGetLastError());
    return NULL;
}

extern "C" const char* initRow(int blockSize, int numBlocks, int seqALen)
{
    hipMalloc( (void**)&d_row, sizeof(float) * numBlocks * blockSize * seqALen );
    if ( hipGetLastError() != hipSuccess )
    	return hipGetErrorString(hipGetLastError());
    return NULL;
}

extern "C" const char* initScores(int blockSize, int numBlocks)
{
    hipMalloc( (void**)&d_scores, sizeof(float) * numBlocks * blockSize );
    if ( hipGetLastError() != hipSuccess )
    	return hipGetErrorString(hipGetLastError());
    return NULL;
}

extern "C" long getTotalGlobalMemory()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, currentDeviceNum);
    return deviceProp.totalGlobalMem;
}

extern "C" int getMaxGridBlocks()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, currentDeviceNum);
    return deviceProp.maxGridSize[0];
}

extern "C" void runAlignKernel(int blocks, int blockSize, int seqALen, int seqBLen, float gap)
{
    align<<<blocks, blockSize>>>(d_seqA, d_scores, d_row, seqALen, seqBLen, gap);
}

extern "C" void runSumKernel(int blocks, int blockSize, int smSize, float& sum)
{
	fprintf(stderr,"Making %d blocks with a block size of %d and a shared memory size of %d(%d)\n",blocks,blockSize,smSize,sizeof(float));
	reduceSum1<<<blocks, blockSize, smSize>>>(d_scores, &sum);
	//reduceSum1<<<blocks, blockSize>>>(d_scores, &sum);
}


